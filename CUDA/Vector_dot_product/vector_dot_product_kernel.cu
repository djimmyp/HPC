#include "hip/hip_runtime.h"
#ifndef _VECTOR_DOT_PRODUCT_KERNEL_H_
#define _VECTOR_DOT_PRODUCT_KERNEL_H_

#define BLOCK_SIZE 32
#define GRID_SIZE 1280

texture<float> A_on_tex;
texture<float> B_on_tex;

__global__ void vector_dot_product_kernel(float* P, const float* A, const float* B, int num_elements)
{
		__shared__ float thread_sums[ BLOCK_SIZE ];
		float local_sum = 0;
		unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;
		unsigned int pitch = blockDim.x * gridDim.x;
		int i;
		for(int z = tid; z < num_elements; z+= pitch)
		{
			float A_element = tex1Dfetch(A_on_tex, (z));
			float B_element = tex1Dfetch(B_on_tex, (z));
			local_sum += A_element * B_element;
		}
		thread_sums[threadIdx.x] = local_sum;
		__syncthreads();

	/* Reduction performed in each block */
	i = BLOCK_SIZE / 2;   	
	while ( i != 0 ) 
	{
		
		if ( threadIdx.x < i ) {
		
			thread_sums[threadIdx.x] += thread_sums[ threadIdx.x + i ];
		}
		__syncthreads();

		i = i / 2;
	}

	if(threadIdx.x == 0)
	{
		atomicAdd(P, thread_sums[0]);
	}		

}

#endif // #ifndef _VECTOR_DOT_PRODUCT_KERNEL_H
